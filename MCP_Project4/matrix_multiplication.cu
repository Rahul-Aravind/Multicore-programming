#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
//int N = 0;

__global__
void matrix_mult_1(int *a, int ra, int ca, int *b, int rb, int cb, int *c )
{
   
   for( int i = 0;i<ra;i++)
   {
		for(int j=0;j<cb;j++)
		{
		
		   int prod = a[i * ca + threadIdx.x] * b[threadIdx.x * cb + j];
		   atomicAdd(c + (i * cb + j), prod);
		}
   }
}

__global__
void matrix_mult_2(int *a, int ra, int ca, int *b, int rb, int cb, int *c )
{
   
   for(int k = 0;k<ca;k++)
   {
		c[blockIdx.x * cb + blockIdx.y] += a[blockIdx.x * ca + k] * b[k * cb + blockIdx.y]; 
   }
}

void cpu_matrix_multiplication(int *A, int *B, int *C, int ra, int ca, int rb, int cb)
{
	for(int i=0 ; i < ra;i++)
	{
		for( int j=0; j < cb; j++)
		{
			C[i*cb + j] = 0;
			for(int k=0; k<ca;k++)
			{
				C[i*cb + j] += A[i*ca + k] * B[ k * cb + j];
			}
		}
	}
} 
 
 
 
void fillMatrix(int *mat, int rows, int columns)
{
	for( int i = 0; i < rows; i++ )
    {
		for(int j = 0; j < columns ; j++)
		{
			mat[i*columns + j] = rand( ) % 10;
		}
    }
}
 
void printMatrix(int *mat,int rows, int columns)
{
	for( int i = 0; i < rows; i++ )
    {
		for(int j = 0; j < columns ; j++)
		{
			printf("%d ",mat[i*columns + j]);
		}
		printf("\n");
    }
}

int verifyGPUMultiplication(int *c,int *d, int rows, int columns)
{
	for( int i = 0; i < rows; i++ )
    {
		for(int j = 0; j < columns ; j++)
		{
			if(c[i*columns + j] != d[i*columns + j])
			{
				printf("C: %d, D: %d ",c[i*columns + j], d[i*columns + j]);
				return 0;
			}
		}
	}
	return 1;
}
 
int main( int argc, char *argv[] )
{

   int *a, *b, *c, *d,*e;			// host copies of a, b, c
   int *dev_a, *dev_b, *dev_c,*dev_d;		// device copies of a, b, c
   
   /*  My Code */
	//int size = N * sizeof( int );	// space for N integers

	if(argc < 5) {
      printf("The number of arguments is insufficient.");
      return -1;
   }

   
   // dimensions of the matrices
	int ra = atoi(argv[1]);
	int ca = atoi(argv[2]);
	
	//N = ca;
	
	int a_size = ra * ca * sizeof(int);
	
	int rb = atoi(argv[3]);
	int cb = atoi(argv[4]);
	
	int b_size = rb * cb * sizeof(int);
	
	int c_size = ra * cb; 
   
   
   

   // allocate host copies of a, b, c
   a = ( int * ) malloc( a_size );
   b = ( int * ) malloc( b_size );
   c = ( int * ) calloc( c_size, sizeof(int));
   d = ( int * ) calloc( c_size, sizeof(int));
   e = ( int * ) calloc( c_size, sizeof(int));

   // allocate device copies of a, b, c
   hipMalloc( ( void** ) &dev_a, a_size );
   hipMalloc( ( void** ) &dev_b, b_size );
   hipMalloc( ( void** ) &dev_c, c_size * sizeof(int) );
   hipMalloc( ( void** ) &dev_d, c_size * sizeof(int) );

   // initialize host copies of a, b
   fillMatrix(a,ra,ca);
   fillMatrix(b,rb,cb);

   /*
   //printing Matrices
   printf("Matrix A: \n");
   printMatrix(a,ra,ca);
   printf("\n");
   
   printf("Matrix B: \n");
   printMatrix(b,rb,cb);
   printf("\n");
   */
   // copy inputs to device
   hipMemcpy( dev_a, a, a_size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, b_size, hipMemcpyHostToDevice );

   // launch add( ) kernel on GPU, passing parameters
	
   int start = clock();
   matrix_mult_1<<< 1,ca>>>(dev_a, ra, ca, dev_b, rb, cb, dev_c );    
   int end = clock();
   printf("Time taken for GPU Multiplication flavor 1: %d seconds ",(end-start)/CLOCKS_PER_SEC);
   printf("\n");
   
   
   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, c_size * sizeof(int) , hipMemcpyDeviceToHost );
   
   
   dim3 blocks(ra, cb);
   start = clock();
   matrix_mult_2<<< blocks,1>>>(dev_a, ra, ca, dev_b, rb, cb, dev_d );    
   end = clock();
   printf("Time taken for GPU Multiplication flavor 2: %d seconds ",(end-start)/CLOCKS_PER_SEC);
   printf("\n");
   
   
   // copy device result back to host copy of c
   hipMemcpy( d, dev_d, c_size * sizeof(int) , hipMemcpyDeviceToHost );
   
   
   start = clock();
   cpu_matrix_multiplication(a,b,e,ra,ca,rb,cb);
   end = clock();
   printf("Time taken for CPU Multiplication : %d seconds ",(end-start)/CLOCKS_PER_SEC);
   printf("\n");
   
   if (verifyGPUMultiplication(c,e,ra,cb)==1)
   {
	printf("GPU multiplication flavor 1 is performed without any error\n");
   }
   else
   {
	printf("GPU multiplication has some error(s)\n");
   }
   
   
   if (verifyGPUMultiplication(d,e,ra,cb)==1)
   {
	printf("GPU multiplication flavor 2 is performed without any error\n");
   }
   else
   {
	printf("GPU multiplication has some error(s)\n");
   }
   
   // deallocate host copies of a, b, c
   free( a );
   free( b );
   free( c );
   free( d );
   free(e);

   // deallocate device copies of a, b, c
   hipFree( dev_a ); 
   hipFree( dev_b );
   hipFree( dev_c );
   hipFree( dev_d );

   return 0;

}
